#include "hip/hip_runtime.h"
// LAB 1
#include <wb.h>

__global__ void vecAdd(float *in1, float *in2, float *out, int len) {
  //@@ Insert code to implement vector addition here
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < len) {
    out[i] = in1[i] + in2[i];
  }
}

int main(int argc, char **argv) {
  wbArg_t args;
  int inputLength;
  float *hostInput1;
  float *hostInput2;
  float *hostOutput;
  float *deviceInput1;
  float *deviceInput2;
  float *deviceOutput;

  int deviceCount;
  int size;
  hipGetDeviceCount(&deviceCount);

  args = wbArg_read(argc, argv);
  //@@ Importing data and creating memory on host
  hostInput1 =
      (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  hostInput2 =
      (float *)wbImport(wbArg_getInputFile(args, 1), &inputLength);
  hostOutput = (float *)malloc(inputLength * sizeof(float));

  wbLog(TRACE, "The input length is ", inputLength);

  size = inputLength * sizeof(float);

  //@@ Allocate GPU memory here
  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
  }

  //@@ Copy memory to the GPU here
  hipMalloc((void **) &deviceInput1, size);
  hipMemcpy(deviceInput1, hostInput1, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &deviceInput2, size);
  hipMemcpy(deviceInput2, hostInput2, size, hipMemcpyHostToDevice);
  hipMalloc((void **) &deviceOutput, size);

  //@@ Initialize the grid and block dimensions here
  dim3 DimGrid(inputLength/256, 1, 1);
  if (0 != (inputLength % 256)) { DimGrid.x++; }
  dim3 DimBlock(256, 1, 1);

  //@@ Launch the GPU Kernel here to perform CUDA computation
  vecAdd<<<DimGrid, DimBlock>>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

  hipDeviceSynchronize();
  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostOutput, deviceOutput, size, hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(deviceInput1);
  hipFree(deviceInput2);
  hipFree(deviceOutput);

  wbSolution(args, hostOutput, inputLength);

  free(hostInput1);
  free(hostInput2);
  free(hostOutput);

  return 0;
}
