#include "hip/hip_runtime.h"
// MP5 Reduction
// Input: A num list of length n
// Output: Sum of the list = list[0] + list[1] + ... + list[n-1];

#include <wb.h>

#define BLOCK_SIZE 512 //@@ This value is not fixed and you can adjust it according to the situation

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)
  
__global__ void total(float *input, float *output, int len) {
  //@@ Load a segment of the input vector into shared memory
  //@@ Traverse the reduction tree
  //@@ Write the computed sum of the block to the output vector at the correct index
  __shared__ float partialSum[2*BLOCK_SIZE];
  int t = threadIdx.x;
  int start = 2*blockIdx.x*BLOCK_SIZE;
  if (start+t < len) {
    partialSum[threadIdx.x] = input[start+t];
  } else {
    partialSum[threadIdx.x] = 0.0;
  }
  if (start+t+BLOCK_SIZE < len) {
    partialSum[threadIdx.x+BLOCK_SIZE] = input[start+t+BLOCK_SIZE];
  } else {
    partialSum[threadIdx.x+BLOCK_SIZE] = 0.0;
  }
  for (int stride = BLOCK_SIZE; stride >= 1; stride >>= 1) {
    __syncthreads();
    if (threadIdx.x < stride) {
      partialSum[threadIdx.x] += partialSum[threadIdx.x+stride];
    }
  }
  output[blockIdx.x] = partialSum[0];
}

int main(int argc, char **argv) {
  int ii;
  wbArg_t args;
  float *hostInput;  // The input 1D list
  float *hostOutput; // The output list
  //@@ Initialize device input and output pointers
  float *deviceInput;
  float *deviceOutput;

  int numInputElements;  // number of elements in the input list
  int numOutputElements; // number of elements in the output list

  args = wbArg_read(argc, argv);

  //Import data and create memory on host
  hostInput =
      (float *)wbImport(wbArg_getInputFile(args, 0), &numInputElements);

  numOutputElements = numInputElements / (BLOCK_SIZE << 1);
  if (numInputElements % (BLOCK_SIZE << 1)) {
    numOutputElements++;
  }
  hostOutput = (float *)malloc(numOutputElements * sizeof(float));

  // The number of input elements in the input is numInputElements
  // The number of output elements in the input is numOutputElements

  //@@ Allocate GPU memory
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
  }

  //@@ Copy input memory to the GPU
  hipMalloc((void **) &deviceInput, numInputElements * sizeof(float));
  hipMemcpy(deviceInput, hostInput, numInputElements * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **) &deviceOutput, numOutputElements * sizeof(float));

  //@@ Initialize the grid and block dimensions here
  dim3 dimBlock(BLOCK_SIZE, 1, 1);
  dim3 dimGrid(numOutputElements, 1, 1);

  //@@ Launch the GPU Kernel and perform CUDA computation
  total<<<dimGrid, dimBlock>>>(deviceInput, deviceOutput, numInputElements);
  
  hipDeviceSynchronize();  
  //@@ Copy the GPU output memory back to the CPU
  hipMemcpy(hostOutput, deviceOutput, numOutputElements * sizeof(float), hipMemcpyDeviceToHost);
  
  /********************************************************************
   * Reduce output vector on the host
   * NOTE: One could also perform the reduction of the output vector
   * recursively and support any size input. 
   * For simplicity, we do not require that for this lab.
   ********************************************************************/
  for (ii = 1; ii < numOutputElements; ii++) {
    hostOutput[0] += hostOutput[ii];
  }

  //@@ Free the GPU memory
  hipFree(deviceInput);
  hipFree(deviceOutput);

  wbSolution(args, hostOutput, 1);

  free(hostInput);
  free(hostOutput);

  return 0;
}


