#include "hip/hip_runtime.h"
// LAB 2 FA24

#include <wb.h>

#define wbCheck(stmt)                                                     \
  do {                                                                    \
    hipError_t err = stmt;                                               \
    if (err != hipSuccess) {                                             \
      wbLog(ERROR, "Failed to run stmt ", #stmt);                         \
      wbLog(ERROR, "Got CUDA error ...  ", hipGetErrorString(err));      \
      return -1;                                                          \
    }                                                                     \
  } while (0)


// Compute C = A * B
__global__ void matrixMultiply(float *A, float *B, float *C, int numARows,
                               int numAColumns, int numBRows,
                               int numBColumns, int numCRows,
                               int numCColumns)
{
  //@@ Implement matrix multiplication kernel here
  int Row = blockIdx.y * blockDim.y + threadIdx.y;
  int Col = blockIdx.x * blockDim.x + threadIdx.x;
  if ((Row < numARows) && (Col < numBColumns)) {
    float Pvalue = 0;
    for (int k = 0; k < numBRows; ++k) {
      Pvalue += A[Row*numAColumns+k] * B[k*numBColumns+Col];
    }
    C[Row*numBColumns+Col] = Pvalue;
  } 
}


int main(int argc, char **argv) {
  wbArg_t args;
  float *hostA; // The A matrix
  float *hostB; // The B matrix
  float *hostC; // The output C matrix
  float *deviceA;
  float *deviceB;
  float *deviceC;

  int numARows;    // number of rows in the matrix A
  int numAColumns; // number of columns in the matrix A
  int numBRows;    // number of rows in the matrix B
  int numBColumns; // number of columns in the matrix B
  int numCRows;    // number of rows in the matrix C (you have to set this)
  int numCColumns; // number of columns in the matrix C (you have to set
                   // this)

  args = wbArg_read(argc, argv);

  //@@ Importing data and creating memory on host
  hostA = (float *)wbImport(wbArg_getInputFile(args, 0), &numARows,
                            &numAColumns);
  hostB = (float *)wbImport(wbArg_getInputFile(args, 1), &numBRows,
                            &numBColumns);
  wbLog(TRACE, "The dimensions of A are ", numARows, " x ", numAColumns);
  wbLog(TRACE, "The dimensions of B are ", numBRows, " x ", numBColumns);
  // wbLog(TRACE, "B[0][0] ", hostB[65], " B[0][1] ", hostB[66]);
  //@@ Set numCRows and numCColumns
  numCRows = numARows;
  numCColumns = numBColumns;

  //@@ Allocate the hostC matrix
  hostC = (float *)malloc(numCRows * numCColumns * sizeof(float));
  int deviceCount;
  hipGetDeviceCount(&deviceCount);
  //@@ Allocate GPU memory here
  for (int dev = 0; dev < deviceCount; dev++) {
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
  }

  //@@ Copy memory to the GPU here
  hipMalloc((void **) &deviceA, numARows * numAColumns * sizeof(float));
  hipMemcpy(deviceA, hostA, numARows * numAColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **) &deviceB, numBRows * numBColumns * sizeof(float));
  hipMemcpy(deviceB, hostB, numBRows * numBColumns * sizeof(float), hipMemcpyHostToDevice);
  hipMalloc((void **) &deviceC, numCRows * numCColumns * sizeof(float));

  //@@ Initialize the grid and block dimensions here
  dim3 dimGrid(ceil((1.0*numBColumns)/4), ceil((1.0*numARows)/4), 1);
  dim3 dimBlock(4, 4, 1);

  //@@ Launch the GPU Kernel here
  matrixMultiply<<<dimGrid, dimBlock>>>(deviceA, deviceB, deviceC, numARows, numAColumns, numBRows, numBColumns, numCRows, numCColumns);
  hipDeviceSynchronize();

  //@@ Copy the GPU memory back to the CPU here
  hipMemcpy(hostC, deviceC, numCRows * numCColumns * sizeof(float), hipMemcpyDeviceToHost);

  //@@ Free the GPU memory here
  hipFree(deviceA);
  hipFree(deviceB);
  hipFree(deviceC);

  wbSolution(args, hostC, numCRows, numCColumns);
  // wbLog(TRACE, "hostC[0][0] ", hostC[0], " hostC[0][1] ", hostC[1]);

  free(hostA);
  free(hostB);
  //@@Free the hostC matrix
  free(hostC);

  return 0;
}


